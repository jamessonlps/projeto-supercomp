#include "hip/hip_runtime.h"
// Geral
#include <iostream>
#include <omp.h>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <algorithm>
#include <vector>
#include <string>
#include <chrono>
#include <random>
#include <map>
#include <bitset>

// Para thrust
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;

struct movie
{
  int id;
  int begin;
  int end;
  int category;
  int duration;
};

struct return_gpu
{
  int num_movies_selected;
  int screen_time;
};

void fetch_movies(vector<movie> &movies, int num_movies)
{
  for (int i = 0; i < num_movies; i++)
  {
    movie new_movie;

    new_movie.id = i;
    cin >> new_movie.begin;
    cin >> new_movie.end;
    cin >> new_movie.category;

    if (new_movie.end < new_movie.begin)
    {
      new_movie.duration = 24 - new_movie.begin + new_movie.end;
    }
    else
    {
      new_movie.duration = new_movie.end - new_movie.begin;
    }

    movies[i] = new_movie;
  }
}

struct customized_operator
{
  int num_movies;
  movie *movies;
  int *max_by_cat;
  int num_categories;

  customized_operator(
      int _num_movies,
      movie *_movies,
      int *_max_by_cat,
      int _num_categories)
      : num_movies(_num_movies),
        movies(_movies),
        max_by_cat(_max_by_cat),
        num_categories(_num_categories){};

  __device__ int operator()(const int &movies_combination_id) const
  { // movies_combination_id means the movies that are selected

    bool time_scheduled[24]; // Store if a time slot is already scheduled
    for (int t = 0; t < 24; t++)
    {
      time_scheduled[t] = false;
    }

    int max_by_cat_copy[26]; // Store the max_by_cat in a vector
    for (int i = 0; i < num_categories; i++)
    {
      max_by_cat_copy[i] = max_by_cat[i];
    }

    int num_movies_added = 0;

    for (int i = 0; i < num_movies; i++)
    {
      movie movie_i = movies[i];

      if (num_movies_added >= 24)
        return -1;

      if ((movies_combination_id & (1 << i)) && (max_by_cat_copy[movie_i.category - 1] > 0))
      {
        if (movie_i.begin > movie_i.end)
        {
          // Check if required slots are available for start interval (begin -> 24)
          for (int t = movie_i.begin; t < 24; t++)
          {
            if (time_scheduled[t])
              return -1;
          }
          // Check if required slots are available for end interval (0 -> end)
          for (int t = 0; t < movie_i.end; t++)
          {
            if (time_scheduled[t])
              return -1;
          }

          // Add movie to slots (begin -> 24)
          for (int t = movie_i.begin; t < 24; t++)
          {
            time_scheduled[t] = true;
          }
          // Add movie to slots (0 -> end)
          for (int t = 0; t < movie_i.end; t++)
          {
            time_scheduled[t] = true;
          }

          num_movies_added++;
          max_by_cat_copy[movie_i.category - 1]--;
        }

        else
        {
          // Check if required slots are available for interval (begin -> end)
          for (int t = movie_i.begin; t < movie_i.end; t++)
          {
            if (time_scheduled[t])
              return -1;
          }

          // Add movie to slots (begin -> end)
          for (int t = movie_i.begin; t < movie_i.end; t++)
          {
            time_scheduled[t] = true;
          }

          num_movies_added++;
          max_by_cat_copy[movie_i.category - 1]--;
        }
      }
    }

    return num_movies_added;
  }
};

/**
 * @brief Dynamic programming algorithm for GPU. How this works:
 *
 * A vector with all movies is created (movies_gpu).
 *
 * Then, a vector with all possible movie combinations is created: movie_combinations_gpu.
 * This vector is filled with the number of movies that can be scheduled for each combination.
 *
 * The number of movies that can be scheduled for each combination is calculated by the unary operator customized_operator.
 *
 * @param movies A vector with all movies
 * @param max_by_cat A map with the maximum number of movies per category
 * @param num_categories Number of categories
 * @param num_movies Number of movies
 */
void dynamic_program_gpu(vector<movie> &movies, vector<int> &max_by_cat, int num_categories, int num_movies, return_gpu &solution)
{
  unsigned long int num_combinations = pow(2, num_movies); // Number of possible combinations

  thrust::device_vector<movie> movies_gpu(movies);                     // Vector with all movies in GPU
  thrust::device_vector<int> max_by_cat_gpu(max_by_cat);               // Vector with max_by_cat in GPU
  thrust::device_vector<int> movie_combinations_gpu(num_combinations); // 2 ^ num_movies

  thrust::counting_iterator<int> counter(0); // 2 ^ num_movies (for movie_combinations_gpu)

  thrust::transform(
      counter,                        // Start of input
      counter + num_combinations,     // End of input
      movie_combinations_gpu.begin(), // Output
      customized_operator(
          num_movies,                                      // Number of movies
          thrust::raw_pointer_cast(movies_gpu.data()),     // Pointer to movies in GPU
          thrust::raw_pointer_cast(max_by_cat_gpu.data()), // Pointer to max_by_cat in GPU
          num_categories                                   // Number of categories
          )                                                // Unary Operator
  );

  // Find the maximum element in movie_combinations_gpu
  auto max_element_it = thrust::max_element(movie_combinations_gpu.begin(), movie_combinations_gpu.end());

  // Calculate the index of the maximum element
  int max_element_index = thrust::distance(movie_combinations_gpu.begin(), max_element_it);

  // Obtain the value of the maximum element (number of movies that can be scheduled)
  int max_element_value = *max_element_it;

  bitset<30> bitset(max_element_index);
  int screen_time = 0;

  for (int i = 0; i < num_movies; i++)
  {
    if (bitset[i])
    {
      screen_time += movies[i].duration;
    }
  }

  solution = {max_element_value, screen_time};
}

int main(int argc, char *argv[])
{
  int num_movies;
  int num_categories;

  cin >> num_movies >> num_categories;

  vector<movie> movies(num_movies);
  vector<int> max_by_category(num_categories);

  for (int i = 0; i < num_categories; i++)
  {
    int limit;
    cin >> limit;
    max_by_category[i] = limit;
  }

  fetch_movies(movies, num_movies);

  return_gpu solution;

  auto start_exec = chrono::high_resolution_clock::now();
  dynamic_program_gpu(movies, max_by_category, num_categories, num_movies, solution);
  auto end_exec = chrono::high_resolution_clock::now();
  auto exec_time = chrono::duration_cast<chrono::milliseconds>(end_exec - start_exec).count();

  // Print number of movies
  cout << num_movies << endl;

  // Print number of categories
  cout << num_categories << endl;

  // Print number of movies selected
  cout << solution.num_movies_selected << endl;

  // Print execution time
  cout << exec_time << endl;

  // Print screen time
  cout << solution.screen_time << endl;

  return 0;
}