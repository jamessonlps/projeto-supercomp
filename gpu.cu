#include "hip/hip_runtime.h"
// Geral
#include <iostream>
#include <omp.h>
#include <cmath>
#include <iomanip>
#include <cstdlib>
#include <algorithm>
#include <vector>
#include <string>
#include <chrono>
#include <random>
#include <map>

// Para thrust
#include <thrust/sort.h>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/counting_iterator.h>

using namespace std;

struct movie
{
  int id;
  int begin;
  int end;
  int category;
  int duration;
};

void fetch_categories(map<int, int> &categories, int num_categories)
{
  for (int i = 0; i < num_categories; i++)
  {
    int limit;
    cin >> limit;
    categories[i + 1] = limit;
  }
}

void fetch_movies(vector<movie> &movies, int num_movies)
{
  for (int i = 0; i < num_movies; i++)
  {
    movie new_movie;

    new_movie.id = i;
    cin >> new_movie.begin;
    cin >> new_movie.end;
    cin >> new_movie.category;

    if (new_movie.end < new_movie.begin)
    {
      new_movie.duration = 24 - new_movie.begin + new_movie.end;
    }
    else
    {
      new_movie.duration = new_movie.end - new_movie.begin;
    }

    movies[i] = new_movie;
  }
}

struct unop
{
  int num_movies;
  movie *movies;
  int *max_by_cat;
  int num_categories;

  unop(int _num_movies, movie *_movies, int *_max_by_cat, int _num_categories)
      : num_movies(_num_movies), movies(_movies), max_by_cat(_max_by_cat), num_categories(_num_categories){};

  __device__ int operator()(const int &movies_combination) const
  { // movies_combination means the movies that are selected

    bool time_scheduled[24];                                                            // Store if a time slot is already scheduled
    vector<int> max_by_cat_copy = vector<int>(max_by_cat, max_by_cat + num_categories); // Copy of max_by_cat

    for (int t = 0; t < 24; t++)
    {
      time_scheduled[t] = false;
    }

    int num_movies_added = 0;

    for (int i = 0; i < num_movies; i++)
    {
      movie movie_i = movies[i];

      if (num_movies_added >= 24)
        return -1;

      if ((movies_combination & (1 << i)) && (max_by_cat_copy[movie_i.category - 1] > 0))
      {
        if (movie_i.begin > movie_i.end)
        {
          // Check if required slots are available for start interval (begin -> 24)
          for (int t = movie_i.begin; t < 24; t++)
          {
            if (time_scheduled[t])
              return -1;
          }
          // Check if required slots are available for end interval (0 -> end)
          for (int t = 0; t < movie_i.end; t++)
          {
            if (time_scheduled[t])
              return -1;
          }

          // Add movie to slots (begin -> 24)
          for (int t = movie_i.begin; t < 24; t++)
          {
            time_scheduled[t] = true;
          }
          // Add movie to slots (0 -> end)
          for (int t = 0; t < movie_i.end; t++)
          {
            time_scheduled[t] = true;
          }

          num_movies_added++;
          max_by_cat_copy[movie_i.category - 1]--;
        }

        else
        {
          // Check if required slots are available for interval (begin -> end)
          for (int t = movie_i.begin; t < movie_i.end; t++)
          {
            if (time_scheduled[t])
              return -1;
          }

          // Add movie to slots (begin -> end)
          for (int t = movie_i.begin; t < movie_i.end; t++)
          {
            time_scheduled[t] = true;
          }

          num_movies_added++;
          max_by_cat_copy[movie_i.category - 1]--;
        }
      }
    }

    return num_movies_added;
  }
};

/**
 * @brief Dynamic programming algorithm for GPU. How this works:
 *
 * A vector with all movies is created (movies_gpu).
 *
 * Then, a vector with all possible movie combinations is created: movie_combinations_gpu.
 * This vector is filled with the number of movies that can be scheduled for each combination.
 *
 * The number of movies that can be scheduled for each combination is calculated by the unary operator unop.
 *
 * @param movies A vector with all movies
 * @param max_by_cat A map with the maximum number of movies per category
 * @param num_categories Number of categories
 */
void dynamic_program_gpu(vector<movie> &movies, map<int, int> &max_by_cat, int num_categories)
{
  thrust::device_vector<movie> movies_gpu(movies.size());                   // num_movies
  thrust::device_vector<int> max_by_cat_gpu(num_categories);                // num_categories
  thrust::device_vector<int> movie_combinations_gpu(pow(movies.size(), 2)); // num_movies ^ 2
  thrust::counting_iterator<int> counter(0);                                // num_movies ^ 2 (for movie_combinations_gpu)

  movies_gpu = movies;

  for (int i = 0; i < num_categories; i++)
  {
    max_by_cat_gpu[i] = max_by_cat[i + 1];
  }

  thrust::transform(
      counter,                                                                                  // Start of input
      counter + pow(movies.size(), 2),                                                          // End of input
      movie_combinations_gpu.begin(),                                                           // Output
      unop(movies.size(), movies_gpu.data().get(), max_by_cat_gpu.data().get(), num_categories) // Unary Operator
  );

  thrust::host_vector<int> movie_combinations_cpu(movie_combinations_gpu.size());
  thrust::copy(movie_combinations_gpu.begin(), movie_combinations_gpu.end(), movie_combinations_cpu.begin());

  int max_movies = 0;
  for (int i = 0; i < movie_combinations_cpu.size(); i++)
  {
    if (movie_combinations_cpu[i] > max_movies)
    {
      max_movies = movie_combinations_cpu[i];
    }
  }

  cout << "Maximum number of movies: " << max_movies << endl;
}

int main(int argc, char *argv[])
{
  int num_movies;
  int num_categories;

  cin >> num_movies >> num_categories;

  vector<movie> movies(num_movies);
  map<int, int> max_by_category;

  fetch_categories(max_by_category, num_categories);
  fetch_movies(movies, num_movies);

  dynamic_program_gpu(movies, max_by_category, num_categories);

  return 0;
}